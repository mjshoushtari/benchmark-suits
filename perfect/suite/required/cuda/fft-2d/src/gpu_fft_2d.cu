/* -*-Mode: C;-*- */

/**BeginCopyright************************************************************
 *
 * $HeadURL: https://pastec.gtri.gatech.edu/svn/svn-dpc/INNC/projects/PERFECT-TAV-ES/suite/wami/kernels/debayer/wami_debayer.c $
 * $Id: wami_debayer.c 8546 2014-04-02 21:36:22Z tallent $
 *
 *---------------------------------------------------------------------------
 * Part of PERFECT Benchmark Suite (hpc.pnnl.gov/projects/PERFECT/)
 *---------------------------------------------------------------------------
 *
 * Copyright ((c)) 2014, Battelle Memorial Institute
 * Copyright ((c)) 2014, Georgia Tech Research Corporation
 * All rights reserved.
 *
 * 1. Battelle Memorial Institute (hereinafter Battelle) and Georgia Tech
 *    Research Corporation (GTRC) hereby grant permission to any person
 *    or entity lawfully obtaining a copy of this software and associated
 *    documentation files (hereinafter "the Software") to redistribute
 *    and use the Software in source and binary forms, with or without
 *    modification.  Such person or entity may use, copy, modify, merge,
 *    publish, distribute, sublicense, and/or sell copies of the
 *    Software, and may permit others to do so, subject to the following
 *    conditions:
 * 
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimers.
 * 
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in
 *      the documentation and/or other materials provided with the
 *      distribution.
 * 
 *    * Other than as used herein, neither the name Battelle Memorial
 *      Institute nor Battelle may be used in any form whatsoever without
 *      the express written consent of Battelle.
 * 
 *      Other than as used herein, neither the name Georgia Tech Research
 *      Corporation nor GTRC may not be used in any form whatsoever
 *      without the express written consent of GTRC.
 * 
 *    * Redistributions of the software in any form, and publications
 *      based on work performed using the software should include the
 *      following citation as a reference:
 * 
 *      Kevin Barker, Thomas Benson, Dan Campbell, David Ediger, Roberto
 *      Gioiosa, Adolfy Hoisie, Darren Kerbyson, Joseph Manzano, Andres
 *      Marquez, Leon Song, Nathan R. Tallent, and Antonino Tumeo.
 *      PERFECT (Power Efficiency Revolution For Embedded Computing
 *      Technologies) Benchmark Suite Manual. Pacific Northwest National
 *      Laboratory and Georgia Tech Research Institute, December 2013.
 *      http://hpc.pnnl.gov/projects/PERFECT/
 *
 * 2. THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *    "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *    LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *    FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL
 *    BATTELLE, GTRC, OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *    INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 *    (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 *    SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
 *    HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 *    STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 *    ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED
 *    OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **EndCopyright*************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>

#define CUDA_SAFE(x) if ( hipSuccess != (x) ) { printf("CUDA CALL FAILED AT %d\n", __LINE__ ); exit(1);}
#define BLOCK_SIZE 128

__global__ void gpu_bit_reverse_01 (float * w, unsigned int N, unsigned int M, unsigned int bits)
{
   //thread ID for given bock size and number of blocks
   int thread_number = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x * blockDim.y
        + (threadIdx.y * blockDim.x) + threadIdx.x;
   if(thread_number >= N * M) return;
   //thread ID for given bock size and number of blocks
   int i = (blockIdx.x * blockDim.x) + threadIdx.x;
   if(i >= M) return;
   int j_n = blockIdx.y;
   if(j_n >= N) return;
   unsigned int s, shift;
   s = sizeof(int) * CHAR_BIT - 1;
   shift = s - bits + 1;
   unsigned int r;
   float t_real, t_imag;
   r = i;
   unsigned int v = i;
   for (v >>= 1; v; v >>= 1)
   {
      r <<= 1;
      r |= v & 1;
      s--;
   }
   r <<= s;
   r >>= shift;
   if (i < r) {
      t_real = w[(2*j_n*M) + 2 * r];
      t_imag = w[(2*j_n*M) + 2 * r + 1];
      w[(2*j_n*M) + 2 * r] = atomicExch(&w[(2*j_n*M) + 2 * i], t_real);
      w[(2*j_n*M) + 2 * r + 1] = atomicExch(&w[(2*j_n*M) + 2 * i + 1], t_imag);
   }
}

__global__ void gpu_bit_reverse_02 (float * w, unsigned int N, unsigned int M, unsigned int bits)
{

   //thread ID for given bock size and number of blocks
   int thread_number = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x * blockDim.y
        + (threadIdx.y * blockDim.x) + threadIdx.x;
   if(thread_number >= N * M) return;
   //thread ID for given bock size and number of blocks
   int i = (blockIdx.x * blockDim.x) + threadIdx.x;
   if(i >= N) return;
   int i_m = blockIdx.y;
   if(i_m >= M) return;
   unsigned int s, shift;
   s = sizeof(int) * CHAR_BIT - 1;
   shift = s - bits + 1;
   unsigned int r;
   float t_real, t_imag;
   r = i;
   unsigned int v = i;
   for (v >>= 1; v; v >>= 1)
   {
      r <<= 1;
      r |= v & 1;
      s--;
   }
   r <<= s;
   r >>= shift;
   if (i < r) {
      t_real = w[2*(M*r+i_m)    ];
      t_imag = w[2*(M*r+i_m) + 1];
      w[2*(M*r+i_m)    ] = atomicExch(&w[2*(M*i+i_m)    ], t_real);
      w[2*(M*r+i_m) + 1] = atomicExch(&w[2*(M*i+i_m) + 1], t_imag);
   }
}

__global__ void gpu_compute_result_01 (float * data, int N, int M, unsigned int transform_length, float s, float s2)
{
   //thread ID for given bock size and number of blocks
   int thread_number = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x * blockDim.y
      + (threadIdx.y * blockDim.x) + threadIdx.x;
   if(thread_number >= N * M) return;
   int i_n = blockIdx.y;
   if(i_n >= N) return;
   //thread ID for given bock size and number of blocks
   int t_id = (blockIdx.x * blockDim.x) + threadIdx.x;
   int b = t_id * ( 2 * transform_length); 
   if(b >= M) return;
   float w_real;
   float w_imag;
   w_real = 1.0f;
   w_imag = 0.0f;
   for (int a = 0; a < transform_length; a++) {
      int i, j;
      float z_real, z_imag;
      float t_real, t_imag;
      i = b + a;
      j = b + a + transform_length;
      z_real = data[2*i_n*M + 2*j  ];
      z_imag = data[2*i_n*M + 2*j+1];
      t_real = w_real * z_real - w_imag * z_imag;
      t_imag = w_real * z_imag + w_imag * z_real;
      // write the result 
      data[2*i_n*M + 2*j  ]  = data[2*i_n*M + 2*i  ] - t_real;
      data[2*i_n*M + 2*j+1]  = data[2*i_n*M + 2*i+1] - t_imag;
      data[2*i_n*M + 2*i  ] += t_real;
      data[2*i_n*M + 2*i+1] += t_imag;
      t_real = w_real - (s * w_imag + s2 * w_real);
      t_imag = w_imag + (s * w_real - s2 * w_imag);
      w_real = t_real;
      w_imag = t_imag;
   }
}

__global__ void gpu_compute_result_02 (float * data, int N, int M, unsigned int transform_length, float s, float s2)
{
   //thread ID for given bock size and number of blocks
   int thread_number = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x * blockDim.y
       + (threadIdx.y * blockDim.x) + threadIdx.x;
   if(thread_number >= N * M) return;
   int i_m = blockIdx.y;
   if(i_m >= M) return;
   //thread ID for given bock size and number of blocks
   int t_id = (blockIdx.x * blockDim.x) + threadIdx.x;
   int b = t_id * ( 2 * transform_length); 
   if(b >= N) return;
   float w_real;
   float w_imag;
   w_real = 1.0f;
   w_imag = 0.0f;
   for (int a = 0; a < transform_length; a++) {
      int i, j;
      float z_real, z_imag;
      float t_real, t_imag;
      i = b + a;
      j = b + a + transform_length;
      z_real = data[2*(M*j+i_m)    ];
      z_imag = data[2*(M*j+i_m) + 1];
      t_real = w_real * z_real - w_imag * z_imag;
      t_imag = w_real * z_imag + w_imag * z_real;
      // write the result 
      data[2*(M*j+i_m)    ]  = data[2*(M*i+i_m)    ] - t_real;
      data[2*(M*j+i_m) + 1]  = data[2*(M*i+i_m) + 1] - t_imag;
      data[2*(M*i+i_m)    ] += t_real;
      data[2*(M*i+i_m) + 1] += t_imag;
      t_real = w_real - (s * w_imag + s2 * w_real);
      t_imag = w_imag + (s * w_real - s2 * w_imag);
      w_real = t_real;
      w_imag = t_imag;
   }	
}


extern "C" int gpu_fft_2d (float * data, unsigned int N, unsigned int M, unsigned int logn, unsigned int logm, int sign)
{
   //allocate memory for variable data
   float *dev_data;
   CUDA_SAFE(hipMalloc(&dev_data, 2 * M *N * sizeof(float)));
   CUDA_SAFE(hipMemcpy(dev_data, data, 2 * M * N * sizeof(float), hipMemcpyHostToDevice));
   //perform 1D FFT on each row
   int num_blocks = (M + BLOCK_SIZE - 1) / BLOCK_SIZE;
   //compute number of blocks for kernel launch
   dim3 grid_dim_01(num_blocks,N,1);
   dim3 block_dim_01(BLOCK_SIZE);
   gpu_bit_reverse_01<<<grid_dim_01,block_dim_01>>>(dev_data, N, M, logm);
   CUDA_SAFE(hipDeviceSynchronize());
   // calculation 
   unsigned int transform_length;
   unsigned int bit;
   transform_length = 1;
   for (bit = 0; bit < logn; bit++) {
      float theta, s, t, s2;
      theta = 1.0 * sign * M_PI / (float) transform_length;
      s = sin (theta);
      t = sin (0.5 * theta);
      s2 = 2.0 * t * t;
      int num_blocks = (M / (2 * transform_length) + BLOCK_SIZE - 1) / BLOCK_SIZE;
      //compute number of blocks for kernel launch
      dim3 grid_dim_01(num_blocks,N,1);
      dim3 block_dim_01(BLOCK_SIZE);
      gpu_compute_result_01<<<grid_dim_01,block_dim_01>>>(dev_data, N, M, transform_length, s, s2);
      CUDA_SAFE(hipDeviceSynchronize());
      transform_length *= 2;
   }
   //end of 1D FFT on each row
   //perform 1D FFT on each column
   num_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
   //compute number of blocks for kernel launch
   dim3 grid_dim_02(num_blocks,M,1);
   dim3 block_dim_02(BLOCK_SIZE);
   gpu_bit_reverse_02<<<grid_dim_02,block_dim_02>>>(dev_data, N, M, logm);
   CUDA_SAFE(hipDeviceSynchronize());
   transform_length = 1;
   for (bit = 0; bit < logn; bit++) {
      float theta, s, t, s2;
      theta = 1.0 * sign * M_PI / (float) transform_length;
      s = sin (theta);
      t = sin (0.5 * theta);
      s2 = 2.0 * t * t;
      int num_blocks = (N / (2 * transform_length) + BLOCK_SIZE - 1) / BLOCK_SIZE;
      //compute number of blocks for kernel launch
      dim3 grid_dim_02(num_blocks,M,1);
      dim3 block_dim_02(BLOCK_SIZE);
      gpu_compute_result_02<<<grid_dim_02,block_dim_02>>>(dev_data, N, M, transform_length, s, s2);
      CUDA_SAFE(hipDeviceSynchronize());
      transform_length *= 2;
   }
   //end of 1D FFT on each column
   //copy variable data back from Device to Host
   CUDA_SAFE(hipMemcpy(data, dev_data, (2 * M *N * sizeof(float)), hipMemcpyDeviceToHost));
   //free device variables
   CUDA_SAFE(hipFree(dev_data));
   return 0;
}

